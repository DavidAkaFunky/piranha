
#include <gtest/gtest.h>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/iterator/transform_iterator.h>
#include <vector>

#include "bitwise.cuh"
#include "CNNConfig.h"
#include "CNNLayer.h"
#include "convolution.cuh"
#include "DeviceData.h"
#include "DeviceBuffer.h"
#include "DeviceBufferView.h"
#include "FCConfig.h"
#include "FCLayer.h"
#include "Functionalities.h"
#include "globals.h"
#include "matrix.cuh"
#include "MaxpoolConfig.h"
#include "MaxpoolLayer.h"
#include "Profiler.h"
#include "ReLUConfig.h"
#include "ReLULayer.h"
#include "RSS.h"
#include "secondary.h"
#include "util.cuh"

extern int partyNum;
extern Profiler func_profiler;

namespace testing {

// Just for testing, not for anything serious.
// https://en.wikipedia.org/wiki/Permuted_congruential_generator
static uint64_t state = 0x4d595df4d0f33173;
static uint64_t const multiplier = 6364136223846793005u;
static uint64_t const increment = 1442695040888963407u;

static uint32_t rotr32(uint32_t x, unsigned r) {
    return x >> r | x << (-r & 31);
}

uint32_t pcg32() {
    uint64_t x = state;
    unsigned count = (unsigned)(x >> 59);

    state = x * multiplier + increment;
    x ^= x >> 18;
    return rotr32((uint32_t)(x >> 27), count);
}

void pcg32_init(uint64_t seed) {
    state = seed + increment;
    pcg32();
}

void leftShift(std::vector<uint32_t> &v, int bits) {
    for (int i = 0; i < v.size(); i++) {
        v[i] <<= bits;
    }
}

void rightShift(std::vector<uint32_t> &v, int bits) {
    for (int i = 0; i < v.size(); i++) {
        v[i] >>= bits;
    }
}

template<typename T, typename I, typename C>
void assertDeviceData(DeviceData<T, I, C> &result, std::vector<float> &expected, bool convertFixed=true) {

    ASSERT_EQ(result.size(), expected.size());
    
    std::vector<float> host_result(result.size());
    copyToHost(result, host_result, convertFixed);

    for(int i = 0; i < host_result.size(); i++) {
        ASSERT_EQ(host_result[i], expected[i]);
    }
}

template<typename T, typename I, typename C>
void assertRSS(RSS<T, I, C> &result, std::vector<float> &expected, bool convertFixed=true) {

    ASSERT_EQ(result.size(), expected.size());

    std::vector<float> host_result(result.size());
    copyToHost(result, host_result, convertFixed);

    for(int i = 0; i < host_result.size(); i++) {
        ASSERT_EQ(host_result[i], expected[i]);
    }
}

TEST(DataTest, DISABLED_DeviceBuffer) {

    DeviceBuffer<uint32_t> d1 = {1, 2, 3};
    DeviceBuffer<uint32_t> d2 = {1, 1, 1};

    d1 += d2;

    //thrust::device_vector<uint32_t> v(10, 1);
    //std::cout << abi::__cxa_demangle(typeid(v.begin()).name(), nullptr, nullptr, nullptr) << std::endl;

    //printDeviceData(d1, "test buffer", false);

    std::vector<float> expected = {2, 3, 4};
    assertDeviceData(d1, expected, false);
}

template<typename T>
using VIterator = thrust::detail::normal_iterator<thrust::device_ptr<T> >;
template<typename T>
using VConstIterator = thrust::detail::normal_iterator<thrust::device_ptr<const T> >;

typedef thrust::transform_iterator<thrust::negate<uint32_t>, VIterator<uint32_t> > TIterator;
typedef thrust::transform_iterator<thrust::negate<uint32_t>, VConstIterator<uint32_t> > TConstIterator;

TEST(DataTest, DISABLED_DeviceBufferView) {

    DeviceBuffer<uint32_t> d1 = {1, 2, 3};
    DeviceBufferView<uint32_t, TIterator, TConstIterator> negated(
        thrust::make_transform_iterator(d1.first(), thrust::negate<uint32_t>()),
        thrust::make_transform_iterator(d1.last(), thrust::negate<uint32_t>())
    );

    d1 += negated;

    std::vector<float> expected = {0, 0, 0};
    assertDeviceData(d1, expected, false);
}

TEST(GPUTest, DISABLED_MatMul) {
    DeviceBuffer<uint32_t> a = {1, 2, 1, 2, 1, 2};  // 2 x 3
    DeviceBuffer<uint32_t> b = {2, 1, 2, 1, 2, 1, 2, 1, 2, 1, 2, 1}; // 3 x 4
    DeviceBuffer<uint32_t> c(8); // 2 x 4

    gpu::matrixMultiplication(a, b, c, false, false, 2, 3, 4);
    hipDeviceSynchronize();

    std::vector<float> expected = {8, 4, 8, 4, 10, 5, 10, 5};
    assertDeviceData(c, expected, false);
}

TEST(GPUTest, DISABLED_MatMulTranspose) {
    DeviceBuffer<uint32_t> a = {1, 2, 1, 2, 1, 2}; // 2 x 3
    DeviceBuffer<uint32_t> c(4); // 2 x 2

    gpu::matrixMultiplication(a, a, c, false, true, 2, 3, 2);
    hipDeviceSynchronize();

    std::vector<float> expected = {6, 6, 6, 9};
    assertDeviceData(c, expected, false);
}

TEST(GPUTest, DISABLED_Transpose) {
    
    DeviceBuffer<uint32_t> a = {1, 2, 3, 4, 5, 6};
    DeviceBuffer<uint32_t> b(a.size());
    gpu::transpose(a, b, 2, 3);
    hipDeviceSynchronize();

    std::vector<float> expected = {1, 4, 2, 5, 3, 6};
    assertDeviceData(b, expected, false);
}

TEST(GPUTest, DISABLED_ElementwiseVectorAdd) {
    
    DeviceBuffer<uint32_t> a = {1, 2, 3, 3, 2, 1}; // 2 x 3

    DeviceBuffer<uint32_t> row_b = {1, 1, 2};
    gpu::elementVectorAdd(a, row_b, true, 2, 3);
    hipDeviceSynchronize();
    
    std::vector<float> expected = {2, 3, 5, 4, 3, 3};
    assertDeviceData(a, expected, false);

    DeviceBuffer<uint32_t> col_b = {2, 3};
    gpu::elementVectorAdd(a, col_b, false, 2, 3);
    hipDeviceSynchronize();

    expected = {4, 5, 7, 7, 6, 6};
    assertDeviceData(a, expected, false);
}

TEST(GPUTest, DISABLED_BitExpand) {

    DeviceBuffer<uint32_t> a = {2, 3, 1};

    DeviceBuffer<uint32_t> abits(a.size() * 32);
    gpu::bitexpand(a, abits);
    hipDeviceSynchronize();

    std::vector<float> expected = {
        0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    };
    assertDeviceData(abits, expected, false);
}

TEST(GPUTest, DISABLED_Im2Row) {

    // 2x3, Din=2
    DeviceBuffer<uint32_t> im = {
        1, 2, 1,
        2, 1, 2,
        1, 2, 3,
        2, 0, 1
    };

    DeviceBuffer<uint32_t> out(12*9);
    gpu::im2row(im, out,
        3, 2, // width, height
        3, // filter size
        2, // Din
        1, 1 // stride, padding
    );
    hipDeviceSynchronize();

    std::vector<float> expected = {
        // im 0  im 1 filter windows
        0, 0, 0, 0, 1, 2, 0, 2, 1,  0, 0, 0, 0, 1, 2, 0, 2, 0,
        0, 0, 0, 1, 2, 1, 2, 1, 2,  0, 0, 0, 1, 2, 3, 2, 0, 1,
        0, 0, 0, 2, 1, 0, 1, 2, 0,  0, 0, 0, 2, 3, 0, 0, 1, 0,
        0, 1, 2, 0, 2, 1, 0, 0, 0,  0, 1, 2, 0, 2, 0, 0, 0, 0,
        1, 2, 1, 2, 1, 2, 0, 0, 0,  1, 2, 3, 2, 0, 1, 0, 0, 0,
        2, 1, 0, 1, 2, 0, 0, 0, 0,  2, 3, 0, 0, 1, 0, 0, 0, 0
    };
    assertDeviceData(out, expected, false);
}

TEST(GPUTest, DISABLED_ExpandCompare) {

    RSS<uint32_t, DeviceVectorIterator<uint32_t>, DeviceVectorConstIterator<uint32_t> > b = {0, 1};
    b >>= FLOAT_PRECISION;
    RSS<uint32_t, DeviceVectorIterator<uint32_t>, DeviceVectorConstIterator<uint32_t> > negb = {1, 0};
    negb >>= FLOAT_PRECISION;
    RSS<uint32_t, DeviceVectorIterator<uint32_t>, DeviceVectorConstIterator<uint32_t> > output(4);

    gpu::expandCompare(b, negb, output);
    hipDeviceSynchronize();

    std::vector<float> expected = {0, 1, 1, 0};
    assertRSS(output, expected, false);
}

TEST(FuncTest, DISABLED_Reconstruct2of3) {
    
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t>> a = {1, 2, 3, 10, 5};

    DeviceBuffer<uint32_t> r(a.size());
    NEW_funcReconstruct(a, r);

    std::vector<float> expected = {1, 2, 3, 10, 5};
    assertDeviceData(r, expected);
}

TEST(FuncTest, DISABLED_Reconstruct3of3) {
    DeviceBuffer<uint32_t> *data;
    switch (partyNum) {
        case PARTY_A:
            data = new DeviceBuffer<uint32_t>({1, 2, 3, 4});
            break;
        case PARTY_B:
            data = new DeviceBuffer<uint32_t>({1, 1, 2, 2});
            break;
        case PARTY_C:
            data = new DeviceBuffer<uint32_t>({1, 1, 0, 0});
            break;
    }

    DeviceBuffer<uint32_t> r(data->size());
    NEW_funcReconstruct3out3(*data, r);

    std::vector<float> expected = {3, 4, 5, 6};
    assertDeviceData(r, expected, false);
}

TEST(FuncTest, DISABLED_MatMul) {

    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > a = {1, 1, 1, 1, 1, 1};  // 2 x 3
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > b = {1, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1, 0}; // 3 x 4
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > c(8); // 2 x 4

    NEW_funcMatMul(a, b, c, 2, 3, 4, false, false, FLOAT_PRECISION);

    std::vector<float> expected = {1, 1, 1, 0, 1, 1, 1, 0};
    assertRSS(c, expected);
}

TEST(FuncTest, DISABLED_Reshare) {
    
    DeviceBuffer<uint32_t> *a;
    if (partyNum == PARTY_A) {
        a = new DeviceBuffer<uint32_t>({1, 2, 3, 4});
    } else {
        a = new DeviceBuffer<uint32_t>(4);
        a->zero();
    } 
       
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > reshared(a->size());
    NEW_funcReshare(*a, reshared);

    std::vector<float> expected = {1, 2, 3, 4};
    assertRSS(reshared, expected, false);
}

TEST(FuncTest, DISABLED_SelectShare) {

    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > x = {1, 2};
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > y = {4, 5};
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > b = {0, 1};
    *b[0] >>= FLOAT_PRECISION;
    *b[1] >>= FLOAT_PRECISION;

    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > z(x.size());
    NEW_funcSelectShare(x, y, b, z);

    std::vector<float> expected = {1, 5};
    assertRSS(z, expected);
}

TEST(FuncTest, DISABLED_Truncate) {

    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > a = {1 << 3, 2 << 3, 3 << 3};
    NEW_funcTruncate(a, 3);

    std::vector<float> expected = {1, 2, 3};
    assertRSS(a, expected);
}

TEST(FuncTest, DISABLED_Convolution) {

    // 2x3, Din=2
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > im = {
        1, 2, 1,
        2, 1, 2,
        1, 2, 3,
        2, 0, 1
    };

    // 2 3x3 filters, Dout=1
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > filters = {
        1, 1, 1,
        1, 1, 1,
        1, 1, 1,
        1, 0, 0,
        0, 1, 0,
        0, 0, 1
    };

    // 1xDout, duplicated for each row in the convolved output
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > biases = {
        1
    };

    // imageW - filterSize + (2*padding) / stride + 1
    size_t wKernels = ((3 - 3 + (2 * 1))/1)+1;
    // imageH - filterSize + (2*padding) / stride + 1
    size_t hKernels = ((2 - 3 + (2 * 1))/1)+1;
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > out(wKernels * hKernels * 1); // Dout = 1

    NEW_funcConvolution(im, filters, biases, out,
            3, 2, 3, 2, 1, 1, 1, FLOAT_PRECISION);

    std::vector<float> expected = {
        8, 13, 10, 9, 11, 10
    };
    assertRSS(out, expected);
}

TEST(FuncTest, DISABLED_CarryOut) {
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > p = {0, 1, 0, 1, 0, 1, 0, 1};
    p >>= FLOAT_PRECISION;
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > g = {0, 1, 0, 1, 0, 1, 0, 1};
    g >>= FLOAT_PRECISION;

    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > out(2);
    carryOut(p, g, 4, out);

    std::vector<float> expected = {1, 1};
    assertRSS(out, expected, false);
}

TEST(FuncTest, DISABLED_DRELU) {
    
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > input = {-1, 2, -2, -3};

    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > result(input.size());
    NEW_funcDRELU(input, result);

    std::vector<float> expected = {
        0, 1, 0, 0
    };
    assertRSS(result, expected, false);
}

TEST(FuncTest, DISABLED_RELU) {

    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > input = {
        -2, -3, 4, 3, 3.5, 1, -1.5, -1
    };

    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > result(input.size());
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > dresult(input.size());
    NEW_funcRELU(input, result, dresult);

    std::vector<float> expected = {
        0, 0, 4, 3, 3.5, 1, 0, 0
    };
    assertRSS(result, expected);

    std::vector<float> dexpected = {
        0, 0, 1, 1, 1, 1, 0, 0
    };
    assertRSS(dresult, dexpected, false);
}

TEST(FuncTest, DISABLED_Maxpool) {

    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > input = {1, 3, 4, 3, 7, 1, 2, 10};
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > result(input.size() / 4);
    RSS<uint32_t, VIterator<uint32_t>, VConstIterator<uint32_t> > dresult(input.size());

    NEW_funcMaxpool(input, result, dresult, 4);

    std::vector<float> expected = {
        4, 10
    };
    assertRSS(result, expected);

    std::vector<float> dexpected = {
        0, 0, 1, 0, 0, 0, 0, 1
    };
    assertRSS(dresult, dexpected, false);
}

TEST(LayerTest, DISABLED_FCForward) {

    int inputDim = 4;
    int batchSize = 4;
    int outputDim = 3;

    RSS<uint32_t, DeviceVectorIterator<uint32_t>, DeviceVectorConstIterator<uint32_t> > input {
        1, 0, 0, 0,
        0, 1, 0, 0,
        0, 0, 1, 0,
        0, 0, 0, 1
    };

    FCConfig *lconfig = new FCConfig(inputDim, batchSize, outputDim);
    FCLayer<uint32_t, DeviceVectorIterator<uint32_t>, DeviceVectorConstIterator<uint32_t> > layer(lconfig, 0); 

    layer.forward(input);

    std::vector<float> expected(4*3); // 12
    copyToHost(*(layer.getWeights()), expected);

    assertRSS(*layer.getActivation(), expected);
}

TEST(LayerTest, CNNForward) {

    //DeviceBuffer<uint32_t>::printMemUsage();

    // 2x2, Din=3
    //std::cout << "allocating im RSS" << std::endl;
    RSS<uint32_t, DeviceVectorIterator<uint32_t>, DeviceVectorConstIterator<uint32_t> > im = {
        1, 0,
        0, 0,
        0, 1,
        0, 0,
        0, 0,
        0, 1
    };

    // weights: 2 2x2 filters, Dout=2 -> 4 2x2 filters
    // biases: 1xDout, duplicated for each row in the convolved output
    //              -> 1x2 biases
    //std::cout << "allocating CNN config" << std::endl;
    CNNConfig *lconfig = new CNNConfig(
        2, 2, // image width x image height
        3, // input features
        2, 2, // filters, filter size
        2, 1, // stride, padding
        1 // batch size
    );
    //std::cout << "allocating CNN layer" << std::endl;
    CNNLayer<uint32_t, DeviceVectorIterator<uint32_t>, DeviceVectorConstIterator<uint32_t> > layer(lconfig, 0); 

    layer.forward(im);

    // construct expected results based on randomized layer weights 
    std::vector<float> host_weights(2*2*3*2); // 24
    //std::cout << "copy to host" << std::endl;
    copyToHost(*layer.getWeights(), host_weights);

    std::vector<float> expected = {
        host_weights[3],
        host_weights[6],
        0,
        host_weights[8],
        host_weights[15],
        host_weights[18],
        0,
        host_weights[20]
    };

    //std::cout << "assert RSS" << std::endl;
    assertRSS(*layer.getActivation(), expected);
}

TEST(LayerTest, DISABLED_CNNPerf) {

    RSS<uint32_t, DeviceVectorIterator<uint32_t>, DeviceVectorConstIterator<uint32_t> > im(64 * 64 * 64);
    im.fill(1);

    // weights: 2 2x2 filters, Dout=2 -> 4 2x2 filters
    // biases: 1xDout, duplicated for each row in the convolved output
    //              -> 1x2 biases
    CNNConfig *lconfig = new CNNConfig(
        64, 64, // image width x image height
        64, // input features
        64, 3, // filters, filter size
        1, 1, // stride, padding
        128 // batch size
    );
    CNNLayer<uint32_t, DeviceVectorIterator<uint32_t>, DeviceVectorConstIterator<uint32_t> > layer(lconfig, 0); 
    Profiler p;
    p.start();
    layer.forward(im);
    p.accumulate("CNN forward");
    p.dump_all();
}

TEST(LayerTest, DISABLED_RELUForward) {

    RSS<uint32_t, DeviceVectorIterator<uint32_t>, DeviceVectorConstIterator<uint32_t> > input = {
        -2, -3, 4, 3, 3.5, 1, -1.5, -1
    };

    ReLUConfig *lconfig = new ReLUConfig(
        input.size(),
        1 // batch size? 
    );
    ReLULayer<uint32_t, DeviceVectorIterator<uint32_t>, DeviceVectorConstIterator<uint32_t> > layer(lconfig, 0);
    layer.forward(input);

    std::vector<float> expected = {
        0, 0, 4, 3, 3.5, 1, 0, 0
    };
    assertRSS(*layer.getActivation(), expected);
}

TEST(LayerTest, DISABLED_MaxpoolForward) {
    // imageWidth x imageHeight = 2 x 2
    // features = 3
    // batchSize = 2
    RSS<uint32_t, DeviceVectorIterator<uint32_t>, DeviceVectorConstIterator<uint32_t> > inputImage = {
        // im 1
         1,  3,
        -3,  0,
        -1, -2,
         2,  1,
         3,  0,
         1, -1,
        // im 2
         2, -2,
         0, -1,
        -1, -3,
         0,  3,
        -2, -3,
         0, -2
    };

    MaxpoolConfig *lconfig = new MaxpoolConfig(
        2, 2, // imageWidth x imageHeight
        3, // features
        2, // poolSize
        1, // stride
        2 // batchSize
    );
    MaxpoolLayer<uint32_t, DeviceVectorIterator<uint32_t>, DeviceVectorConstIterator<uint32_t> > layer(lconfig, 0);
    layer.forward(inputImage);

    std::vector<float> expected = {
        3, 2, 3, 2, 3, 0    
    };
    assertRSS(*layer.getActivation(), expected);
}

/*
TEST(PerfTest, DISABLED_LargeMatMul) {

    int rows = 8;
    int shared = 784; // 786
    int cols = 128; // 128

    RSSData<uint32_t> a(rows * shared);
    RSSData<uint32_t> b(shared *  cols);
    RSSData<uint32_t> c(rows * cols);

    //std::cout << "generating inputs" << std::endl;

    / *
    std::default_random_engine generator;

    std::uniform_int_distribution<uint32_t> distribution(0,255);
    std::vector<uint32_t> randomInput(rows * shared);
    for (int i = 0; i < randomInput.size(); i++) {
        randomInput.push_back(distribution(generator));
    }
    if (partyNum == PARTY_A) {
        thrust::copy(randomInput.begin(), randomInput.end(), a[0].getData().begin());
    } else if (partyNum == PARTY_C) {
        thrust::copy(randomInput.begin(), randomInput.end(), a[1].getData().begin());
    }
    * /

    //std::cout << "generating weights" << std::endl;

    / *
    std::uniform_int_distribution<uint32_t> bit_distribution(0,1);
    std::vector<uint32_t> randomWeights(shared * cols);
    for (int i = 0; i < randomWeights.size(); i++) {
        randomInput.push_back(bit_distribution(generator));
    }
    if (partyNum == PARTY_A) {
        thrust::copy(randomWeights.begin(), randomWeights.end(), b[0].getData().begin());
    } else if (partyNum == PARTY_C) {
        thrust::copy(randomWeights.begin(), randomWeights.end(), b[1].getData().begin());
    }
    * /

    Profiler p;
    p.start();
    NEW_funcMatMul(a, b, c, rows, shared, cols, false, false, FLOAT_PRECISION);
    p.accumulate("matmul");

    p.dump_all();
}

TEST(PerfTest, DISABLED_FCLayer) {

    int inputDim = 784;
    int batchSize = 8;
    int outputDim = 128;

    / *
    std::default_random_engine generator;

    std::uniform_int_distribution<uint32_t> distribution(0,255);
    std::vector<uint32_t> randomVals(batchSize * inputDim);
    for (int i = 0; i < randomVals.size(); i++) {
        randomVals.push_back(distribution(generator));
    }* /

    RSSData<uint32_t> input(batchSize * inputDim);
    / *
    if (partyNum == PARTY_A) {
        thrust::copy(randomVals.begin(), randomVals.end(), input[0].getData().begin());
    } else if (partyNum == PARTY_C) {
        thrust::copy(randomVals.begin(), randomVals.end(), input[1].getData().begin());
    }
    * /

    FCConfig *lconfig = new FCConfig(inputDim, batchSize, outputDim);
    FCLayer<uint32_t> layer(lconfig, 0); 

    layer.forward(input);
    layer.layer_profiler.dump_all();
}

TEST(PerfTest, DISABLED_ReLULayer) {

    func_profiler.clear();

    int inputDim = 1000000;
    int batchSize = 1;

    RSSData<uint32_t> input(batchSize * inputDim);

    ReLUConfig *lconfig = new ReLUConfig(inputDim, batchSize);
    ReLULayer<uint32_t> layer(lconfig, 0); 

    start_m();

    layer.forward(input);

    end_m("relu-test");

	cout << "----------------------------------------------" << endl;  	
	cout << "Run details: " << NUM_OF_PARTIES << "PC (P" << partyNum 
		 << "), " << NUM_ITERATIONS << " iterations, batch size " << MINI_BATCH_SIZE << endl ;
	cout << "----------------------------------------------" << endl << endl;  

    layer.layer_profiler.dump_all();
    func_profiler.dump_all();
}
*/

} // namespace test

int runTests(int argc, char **argv) {
    testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}

